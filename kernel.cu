#include "hip/hip_runtime.h"
#include "kernel.h"
#include "structs.h"
#include <math.h>	
#include <algorithm>
#include <thrust/execution_policy.h>
#include <thrust/binary_search.h>
#include "params.h"


__device__ void swap(unsigned int* a, unsigned int* b) {
	unsigned int temp = *a;
	*a = *b;
	*b= temp;
}

__device__ void sortCell(unsigned int* list, DTYPE* database, int length, int tid){
	bool odd=false;
	for(int i=0; i<length; i++) {
		for(int j=(tid*2)+(int)odd; j<length-1; j+=32) {
			if(database[list[j]*GPUNUMDIM] > database[list[j+1]*GPUNUMDIM]) {
				swap(&list[j], &list[j+1]);
			}
		}
		odd = !odd;
	}
}

__device__ void seqSortCell(unsigned int* list, DTYPE* database, int length){
	int min;
	int minIdx;

	for(int i=0; i<length-1; i++ ) {
		min = database[list[i]*GPUNUMDIM];
		minIdx=i;
		for(int j=i; j<length; i++) {
			if(database[list[j]*GPUNUMDIM] < min) {
				min = database[list[j]*GPUNUMDIM];
				minIdx = j;
			}
		}
		swap(&list[i], &list[minIdx]);
	}
}


__global__ void kernelSortPointsInCells(DTYPE* database, struct grid * index, unsigned int* indexLookupArr, unsigned int nNonEmptyCells) {
        int tid = threadIdx.x + (blockIdx.x*BLOCKSIZE);
        int warpId = tid/32;
        int totalWarps = (gridDim.x*BLOCKSIZE)/32;

	int sortDim=0;
	if(GPUNUMDIM > NUMINDEXEDDIM)
		sortDim = NUMINDEXEDDIM;
	

        for(int i=warpId; i<nNonEmptyCells; i+=totalWarps) {
		if(index[i].indexmin < index[i].indexmax) {
  	              sortCell(indexLookupArr+index[i].indexmin, database+sortDim, (index[i].indexmax-index[i].indexmin)+1, threadIdx.x%32);
		}
        }

}




__device__ uint64_t getLinearID_nDimensionsGPU(unsigned int * indexes, unsigned int * dimLen, unsigned int nDimensions) {
    uint64_t offset = 0;
	uint64_t multiplier = 1;
	for (int i = 0; i<nDimensions; i++){
  	offset += (uint64_t)indexes[i] * multiplier;
  	multiplier *= dimLen[i];
	}

	return offset;
}

__device__ void getNDimIndexesFromLinearIdxGPU(unsigned int * indexes, unsigned int * dimLen, unsigned int nDimensions, uint64_t linearId) {
	// do the process to get linear id but backwards
	for (int i=0; i<nDimensions; i++){
  	indexes[i] = linearId % dimLen[i];
	linearId /= dimLen[i];
	}
}

// Kernel for binary search
// Needed to get the index of the searched value
__device__ int binarySearch(uint64_t* d_array, unsigned int size, uint64_t value) {
    int low = 0;
    int high = size - 1;

    while (low <= high) {
        int mid = low + (high - low) / 2;

		// Check if x is present at mid
        if (d_array[mid] == value) {
            return mid;
		}
        if (d_array[mid] < value) {
			// Move to the right half
            low = mid + 1; 
        } else {
			// Move to the left half
            high = mid - 1;
        }
    }

	// return cell not found
	return -1;
}


//unique key array on the GPU
__global__ void kernelUniqueKeys(int * pointIDKey, unsigned int * N, int * uniqueKey, int * uniqueKeyPosition, unsigned int * cnt)
{
	int tid=threadIdx.x+ (blockIdx.x*BLOCKSIZE); 

	if (tid>=*N){
		return;
	}	

	if (tid==0)
	{
		unsigned int idx=atomicAdd(cnt,int(1));
		uniqueKey[idx]=pointIDKey[0];
		uniqueKeyPosition[idx]=0;
		return;
	
	}
	
	//All other threads, compare to previous value to the array and add
	
	if (pointIDKey[tid-1]!=pointIDKey[tid])
	{
	unsigned int idx=atomicAdd(cnt,int(1));
	uniqueKey[idx]=pointIDKey[tid];
	uniqueKeyPosition[idx]=tid;
	}
	
}





//This version is the same as the batch estimator
//One query point per GPU thread


// unsigned int *debug1, unsigned int *debug2 – ignore, debug values
// unsigned int *N – total GPU threads for the kernel  	
// unsigned int * offset -  This is to offset into every nth data point, e.g., every 100th point calculates its neighbors 
// unsigned int *batchNum - The batch number being executed, used to calculate the point being processed
// DTYPE* database – The points in the database as 1 array
// DTYPE* epsilon – distance threshold
// struct grid * index – each non-empty grid cell is one of these, stores the indices into indexLookupArray that coincide with the data points in the database that are in the cell
// unsigned int * indexLookupArr – array of the size of database, has the indices of the datapoints in the database stored contiguously for each grid cell. each grid index cell references this 	
// struct gridCellLookup * gridCellLookupArr, - lookup array to the grid cells, needed to find if a grid cell exists (this is binary searched). Maps the location of the non-empty grid cells in grid * index to their linearized (1-D) array
// DTYPE* minArr – The minimum “edge” of the grid in each dimension
// unsigned int * nCells –The total number of cells in each dimension (if all were indexed), can compute the spatial extent, with minArr[0]+nCells[0]*epsilon, in the 1st dimension
// unsigned int * cnt – the result set size 	
// unsigned int * nNonEmptyCells – the number of non-empty cells in total, this is the size of the gridCellLookupArr
// int * pointIDKey, int * pointInDistVal - result set to be sorted as key/value pairs

__global__ void kernelNDGridIndexGlobal(unsigned int *debug1, unsigned int *debug2, unsigned int *N,  
	unsigned int * offset, unsigned int * indexGroupOffset, DTYPE* database, DTYPE * epsilon, struct grid * index, unsigned int * indexLookupArr, 
	struct gridCellLookup * gridCellLookupArrStart, struct gridCellLookup * gridCellLookupArrStartEnd, DTYPE* minArr, unsigned int * nCells, 
	unsigned int * orderedIndexPntIDs, unsigned int * cnt, int * pointIDKey, int * pointInDistVal, CTYPE* workCounts)
{

unsigned int tid=threadIdx.x+ (blockIdx.x*BLOCKSIZE);

if (tid>=*N){
	return;
}

/*
//If reordering the queries by the amount of work
#if QUERYREORDER==1
//test refactoring no reordering
// unsigned int pointIdx=tid*(*offset)+(*batchNum);
// unsigned int pointOffset=(GPUNUMDIM)*(tid*(*offset)+(*batchNum));

//the point id in the dataset
unsigned int pointIdx=orderedQueryPntIDs[tid*(*offset)+(*batchNum)]; 
//The offset into the database, taking into consideration the length of each dimension
unsigned int pointOffset=(GPUNUMDIM)*pointIdx; 


#endif

//If standard execution without reordering the queries by the amount of work
#if QUERYREORDER==0
//the point id in the dataset
unsigned int pointIdx=tid*(*offset)+(*batchNum);
//The offset into the database, taking into consideration the length of each dimension
unsigned int pointOffset=(GPUNUMDIM)*pointIdx;
#endif
*/

//the point id in the dataset
unsigned int pointIdx=orderedIndexPntIDs[tid*(*offset) + (*indexGroupOffset)]; 
//The offset into the database, taking into consideration the length of each dimension
unsigned int pointOffset=(GPUNUMDIM)*pointIdx;



//make a local copy of the point
//Store query point in registers
DTYPE point[GPUNUMDIM];
for (int i=0; i<GPUNUMDIM; i++){
	point[i]=database[pointOffset+i];	
}

//calculate the coords of the Cell for the point
//and the min/max ranges in each dimension
unsigned int nDCellIDs[NUMINDEXEDDIM];
unsigned int nDMinCellIDs[NUMINDEXEDDIM];
unsigned int nDMaxCellIDs[NUMINDEXEDDIM];

for (unsigned int i=0; i<NUMINDEXEDDIM; i++){
	nDCellIDs[i]=(point[i]-minArr[i])/(*epsilon);
	nDMinCellIDs[i]=max(0,nDCellIDs[i]-1); //boundary conditions (don't go beyond cell 0)
	nDMaxCellIDs[i]=min(nCells[i]-1,nDCellIDs[i]+1); //boundary conditions (don't go beyond the maximum number of cells)

}
	
        unsigned int indexes[NUMINDEXEDDIM];
        unsigned int loopRng[NUMINDEXEDDIM];

#if STAMP

	for(int i=0; i<NUMINDEXEDDIM; i++) {
		indexes[i] = nDCellIDs[i];
	}
	evaluateCell(nCells, indexes, gridCellLookupArr, nNonEmptyCells, database, epsilon, index, indexLookupArr, point, cnt, pointIDKey, pointInDistVal, pointIdx, false, nDCellIDs, workCounts);

    #include "stamploops.h"

#elif LINEARSTAMP
	for(int i=0; i<NUMINDEXEDDIM; i++) {
		indexes[i] = nDCellIDs[i];
	}
	evaluateCell(nCells, indexes, gridCellLookupArr, nNonEmptyCells, database, epsilon, index, indexLookupArr, point, cnt, pointIDKey, pointInDistVal, pointIdx, false, nDCellIDs, workCounts);

	for (loopRng[0]=rangeFilteredCellIdsMin[0]; loopRng[0]<=rangeFilteredCellIdsMax[0]; loopRng[0]++)
	for (loopRng[1]=rangeFilteredCellIdsMin[1]; loopRng[1]<=rangeFilteredCellIdsMax[1]; loopRng[1]++)
	#include "kernelloops.h"					
	{ //beginning of loop body
	
	for (int x=0; x<NUMINDEXEDDIM; x++){
	indexes[x]=loopRng[x];	
	}
	if(getLinearID_nDimensionsGPU(indexes, nCells, NUMINDEXEDDIM) > getLinearID_nDimensionsGPU(nDCellIDs, nCells, NUMINDEXEDDIM)) {
		evaluateCell(nCells, indexes, gridCellLookupArr, nNonEmptyCells, database, epsilon, index, indexLookupArr, point, cnt, pointIDKey, pointInDistVal, pointIdx, true, nDCellIDs, workCounts);
	}	
	} //end loop body
#else

	for (loopRng[0]=nDMinCellIDs[0]; loopRng[0]<=nDMaxCellIDs[0]; loopRng[0]++)
	for (loopRng[1]=nDMinCellIDs[1]; loopRng[1]<=nDMaxCellIDs[1]; loopRng[1]++)
	#include "kernelloops.h"					
	{ //beginning of loop body
	
	for (int x=0; x<NUMINDEXEDDIM; x++){
	indexes[x]=loopRng[x];	
	}
		evaluateCell(nCells, indexes, database, epsilon, index, indexLookupArr, gridCellLookupArrStart, gridCellLookupArrStartEnd, point, cnt, pointIDKey, pointInDistVal, pointIdx, false, nDCellIDs, workCounts);
	
	} //end loop body
#endif

}

__forceinline__ __device__ void evalPoint(unsigned int* indexLookupArr, int k, DTYPE* database, DTYPE * epsilon, DTYPE* point, 
	unsigned int* cnt, int* pointIDKey, int* pointInDistVal, int pointIdx, bool differentCell) 
{
	
	unsigned int dataIdx=indexLookupArr[k];

	//If we use ILP
	#if ILP>0
	DTYPE runningDist[ILP];
	
	#pragma unroll
	for(int j=0; j<ILP; j++)
		runningDist[j]=0;


	for(int l=0; l<GPUNUMDIM; l+=ILP) {
		#pragma unroll
		for(int j=0; j<ILP && (l+j) < GPUNUMDIM; j++) {
			runningDist[j] += (database[dataIdx*GPUNUMDIM+l+j]-point[l+j])*(database[dataIdx*GPUNUMDIM+l+j]-point[l+j]);
		}
          	#if SHORTCIRCUIT==1
			#pragma unroll
			for(int j=1; j<ILP; j++) {
				runningDist[0] += runningDist[j];
				runningDist[j]=0;
			}
	        	if (sqrt(runningDist[0])>(*epsilon)) {
        	    	  return;
          		}
         	 #endif
	}

	#pragma unroll
	for(int j=1; j<ILP; j++) {
		runningDist[0] += runningDist[j];
	}

	#endif
	//end ILP

	//No ILP
	#if ILP==0
	DTYPE runningTotalDist=0;
    for (int l=0; l<GPUNUMDIM; l++){
      runningTotalDist+=(database[dataIdx*GPUNUMDIM+l]-point[l])*(database[dataIdx*GPUNUMDIM+l]-point[l]);
      #if SHORTCIRCUIT==1
      if (sqrt(runningTotalDist)>(*epsilon)) {
          return;
      }
      #endif
    }
	#endif
	//end no ILP

    	//distance calculation using either ILP or no ILP
    	#if ILP>0
        if (sqrt(runningDist[0])<=(*epsilon)){
        #endif
        #if ILP==0
        if (sqrt(runningTotalDist)<=(*epsilon)){	
        #endif	

		unsigned int idx=atomicAdd(cnt,int(1));
		pointIDKey[idx]=pointIdx;
		pointInDistVal[idx]=dataIdx;
		  
            if(differentCell) {
              unsigned int idx = atomicAdd(cnt,int(1));
			  pointIDKey[idx]=pointIdx;
              pointInDistVal[idx]=dataIdx;
           }
	}
}




__device__ void evaluateCell(unsigned int* nCells, unsigned int* indexes, DTYPE* database, 
	DTYPE * epsilon, struct grid * index, unsigned int * indexLookupArr, struct gridCellLookup * gridCellLookupArrStart, struct gridCellLookup * gridCellLookupArrStartEnd, 
	DTYPE* point, unsigned int* cnt,int* pointIDKey, int* pointInDistVal, int pointIdx, bool differentCell, unsigned int* nDCellIDs, CTYPE* workCounts)
{


#if COUNTMETRICS == 1
			atomicAdd(&workCounts[1],int(1));
#endif

	uint64_t calcLinearID=getLinearID_nDimensionsGPU(indexes, nCells, NUMINDEXEDDIM);

	//compare the linear ID with the gridCellLookupArr to determine if the cell is non-empty: this can happen because one point says 
	//a cell in a particular dimension is non-empty, but that's because it was related to a different point (not adjacent to the query point)

	struct gridCellLookup tmp;
	tmp.gridLinearID=calcLinearID;

	if (thrust::binary_search(thrust::seq, gridCellLookupArrStart, gridCellLookupArrStartEnd, gridCellLookup(tmp))){

        //compute the neighbors for the adjacent non-empty cell
        struct gridCellLookup * resultBinSearch=thrust::lower_bound(thrust::seq, gridCellLookupArrStart, gridCellLookupArrStartEnd, gridCellLookup(tmp));
        unsigned int GridIndex=resultBinSearch->idx;
#if SORT==1
	int sortedDim;
#if GPUNUMDIM == NUMINDEXEDDIM
	sortedDim=0;
	int idx;
	bool mid=false;
	bool left=false;
	if(nDCellIDs[sortedDim] > indexes[sortedDim]){ 
		left = true; 
	}
	else if (nDCellIDs[sortedDim] < indexes[sortedDim]) {
		left =false; 
	}
	else mid = true;

        for(int k=index[GridIndex].indexmin; k<=index[GridIndex].indexmax; k++){
		idx = k;
		if(left) idx = index[GridIndex].indexmax - (k-index[GridIndex].indexmin);
		
		unsigned int dataIdx=indexLookupArr[idx];
		if(std::abs(database[dataIdx*GPUNUMDIM+sortedDim]-point[sortedDim]) > (*epsilon) && !mid) {
			k = index[GridIndex].indexmax+1;
		}
		else {
			evalPoint(indexLookupArr, idx, database, epsilon, point, cnt, pointIDKey, pointInDistVal, pointIdx, differentCell);
#if COUNTMETRICS == 1
			atomicAdd(&workCounts[0],1);
#endif
		}
        }
#else


	sortedDim = NUMINDEXEDDIM;
    // sortedDim = 43; //tmp
	int offset = index[GridIndex].indexmin;
	int length = (index[GridIndex].indexmax - index[GridIndex].indexmin);

	int searchIdx = (length)/2;
	bool lessThan;
	for(int step = (length+3)/4; step > 1; step = (step+1)/2) {
		lessThan = ((point[sortedDim] - database[(indexLookupArr[searchIdx+offset])*GPUNUMDIM+sortedDim]) >= (*epsilon));
		if(lessThan) { 
			searchIdx += step;
			if(searchIdx > length) searchIdx = length;
		}
		else {
			searchIdx -= step;
			if(searchIdx < 0) {
				searchIdx = 0;
				step = 0;
			}
		}
	}
	if(searchIdx > 0 && ((point[sortedDim] - database[indexLookupArr[(searchIdx+offset)-1]*GPUNUMDIM+sortedDim]) <= (*epsilon)))
		searchIdx--;
	if(searchIdx > 0 && ((point[sortedDim] - database[indexLookupArr[(searchIdx+offset)-1]*GPUNUMDIM+sortedDim]) <= (*epsilon)))
		searchIdx--;

	while(searchIdx <= length && ((database[indexLookupArr[searchIdx+offset]*GPUNUMDIM+sortedDim] - point[sortedDim]) < (*epsilon))) {
		evalPoint(indexLookupArr, searchIdx+offset, database, epsilon, point, cnt, pointIDKey, pointInDistVal, pointIdx, differentCell);
		searchIdx++;
#if COUNTMETRICS == 1
			atomicAdd(&workCounts[0],1);
#endif
	}
	
#endif

// Brute force method if SORTED != 1
#else
	for (int k=index[GridIndex].indexmin; k<=index[GridIndex].indexmax; k++){
		evalPoint(indexLookupArr, k, database, epsilon, point, cnt, pointIDKey, pointInDistVal, pointIdx, differentCell);
#if COUNTMETRICS == 1
			atomicAdd(&workCounts[0],1);
#endif
        }
#endif
	}

}



//Kernel brute forces to generate the neighbor table for each point in the database
//cnt with unsigned long long int
__global__ void kernelBruteForce(unsigned int *N, unsigned int *debug1, unsigned int *debug2, DTYPE* epsilon, unsigned long long int * cnt, DTYPE* database, int * pointIDKey, int * pointInDistVal) {

unsigned int tid=threadIdx.x+ (blockIdx.x*BLOCKSIZE); 

if (tid>=*N){
	return;
}


int dataOffset=tid*GPUNUMDIM;
DTYPE runningDist=0;
//compare my point to every other point
for (int i=0; i<(*N); i++)
{
	runningDist=0;
	for (int j=0; j<GPUNUMDIM; j++){
		runningDist+=(database[(i*GPUNUMDIM)+j]-database[dataOffset+j])*(database[(i*GPUNUMDIM)+j]-database[dataOffset+j]);
	}

	//if within epsilon:
	if ((sqrt(runningDist))<=(*epsilon)){
		//with long long unsigned int
		atomicAdd(cnt, (unsigned long long int)1);
		
	}
}


return;
}





//for descriptions of the parameters, see regular kernel that computes the result (not the batch estimator)
__global__ void kernelNDGridIndexBatchEstimator(unsigned int *debug1, unsigned int *debug2, unsigned int *N,  
	unsigned int * sampleOffset, const unsigned int DBSIZE, DTYPE* database, const DTYPE epsilon, unsigned int * whichIndexPoints, struct grid * allIndex, unsigned int * allIndexLookupArr, 
	struct gridCellLookup * allGridCellLookupArr, DTYPE* allMinArr, unsigned int * allNCells, gridCellLookup ** startGridPtrs, gridCellLookup ** stopGridPtrs, grid ** startIndexPtrs,
	unsigned int * cnt, unsigned int * allNNonEmptyCells, unsigned int * orderedQueryPntIDs)
{

unsigned int tid=threadIdx.x+ (blockIdx.x*BLOCKSIZE); 


if (tid>=*N){
	return;
}


//If reordering the queries by the amount of work
#if QUERYREORDER==1
//the point id in the dataset
unsigned int pointIdx=orderedQueryPntIDs[tid*(*sampleOffset)]; 
//The offset into the database, taking into consideration the length of each dimension
unsigned int pointID=(GPUNUMDIM)*pointIdx;
#endif

//If standard execution without reordering the queries by the amount of work
#if QUERYREORDER==0
unsigned int pointIdx = tid*(*sampleOffset);
unsigned int pointID=pointIdx*(GPUNUMDIM);
#endif

//make a local copy of the point
DTYPE point[GPUNUMDIM];
for (int i=0; i<GPUNUMDIM; i++){
	point[i]=database[pointID+i];	
}

unsigned int whichIndex = whichIndexPoints[pointIdx];

//calculate the coords of the Cell for the point
//and the min/max ranges in each dimension
unsigned int nDCellIDs[NUMINDEXEDDIM];
unsigned int nDMinCellIDs[NUMINDEXEDDIM];
unsigned int nDMaxCellIDs[NUMINDEXEDDIM];
for (int i=0; i<NUMINDEXEDDIM; i++){
	nDCellIDs[i]=(point[i]-allMinArr[i + (whichIndex * NUMINDEXEDDIM)])/(epsilon);
	nDMinCellIDs[i]=max(0,nDCellIDs[i]-1); //boundary conditions (don't go beyond cell 0)
	nDMaxCellIDs[i]=min(allNCells[i + (whichIndex * NUMINDEXEDDIM)]-1,nDCellIDs[i]+1); //boundary conditions (don't go beyond the maximum number of cells)

}

	unsigned int indexes[NUMINDEXEDDIM];
	unsigned int loopRng[NUMINDEXEDDIM];

	for (loopRng[0]=nDMinCellIDs[0]; loopRng[0]<=nDMaxCellIDs[0]; loopRng[0]++)
	for (loopRng[1]=nDMinCellIDs[1]; loopRng[1]<=nDMaxCellIDs[1]; loopRng[1]++)
	#include "kernelloops.h"						
	{ //beginning of loop body
	

	#if COUNTMETRICS == 1
			atomicAdd(debug1,int(1));
	#endif	

	for (int x=0; x<NUMINDEXEDDIM; x++){
	indexes[x]=loopRng[x];	
	// if (tid==0)
	// 	printf("\ndim: %d, indexes: %d",x, indexes[x]);
	}
	
	unsigned int nCells[NUMINDEXEDDIM];
	for (int i=0; i<NUMINDEXEDDIM; i++){
		nCells[i] = allNCells[i+(whichIndex*NUMINDEXEDDIM)];
	}

	uint64_t calcLinearID=getLinearID_nDimensionsGPU(indexes, nCells, NUMINDEXEDDIM);

	//compare the linear ID with the gridCellLookupArr to determine if the cell is non-empty: this can happen because one point says 
	//a cell in a particular dimension is non-empty, but that's because it was related to a different point (not adjacent to the query point)

	struct gridCellLookup tmp;
	tmp.gridLinearID=calcLinearID;


	gridCellLookup * startGridPtr = startGridPtrs[whichIndex];
	gridCellLookup * stopGridPtr = stopGridPtrs[whichIndex];
	grid * gridIndexPtr = startIndexPtrs[whichIndex];
	if (thrust::binary_search(thrust::seq, startGridPtr, stopGridPtr, gridCellLookup(tmp))){	
		struct gridCellLookup * resultBinSearch=thrust::lower_bound(thrust::seq, startGridPtr, stopGridPtr, gridCellLookup(tmp));
		unsigned int GridIndex=resultBinSearch->idx;
		
		for (int k=(gridIndexPtr+GridIndex)->indexmin; k<=(gridIndexPtr+GridIndex)->indexmax; k++){
				DTYPE runningTotalDist=0;
				unsigned int dataIdx=allIndexLookupArr[k+(whichIndex*(DBSIZE))];

				

				for (int l=0; l<GPUNUMDIM; l++){
				runningTotalDist+=(database[dataIdx*GPUNUMDIM+l]-point[l])*(database[dataIdx*GPUNUMDIM+l]-point[l]);
				}


				if (sqrt(runningTotalDist)<=(epsilon)){
					//Count number within epsilon
					unsigned int idx=atomicAdd(&cnt[whichIndex],int(1));
				}
			}



	}

	} //end loop body
}

__global__ void kernelInitEnumerateDB(unsigned int * databaseVal, unsigned int *N)
{


	unsigned int tid=threadIdx.x+ (blockIdx.x*BLOCKSIZE); 


	if (tid>=*N){
		return;
	}

	databaseVal[tid]=tid;	

		
}

__global__ void kernelIndexComputeNonemptyCells(DTYPE* database, unsigned int *N, DTYPE* epsilon, DTYPE* minArr, unsigned int * nCells, uint64_t * pointCellArr)
{


	unsigned int tid=threadIdx.x+ (blockIdx.x*BLOCKSIZE); 


	if (tid>=*N){
		return;
	}

	// printf("\n%u",tid); 

	unsigned int pointID=tid*(GPUNUMDIM);

	unsigned int tmpNDCellIdx[NUMINDEXEDDIM];
	for (int j=0; j<NUMINDEXEDDIM; j++){
		tmpNDCellIdx[j]=((database[pointID+j]-minArr[j])/(*epsilon));
	}
	uint64_t linearID=getLinearID_nDimensionsGPU(tmpNDCellIdx, nCells, NUMINDEXEDDIM);

	pointCellArr[tid]=linearID;	

		
}


__global__ void kernelIndexComputeAdjacentCells(uint64_t * celllDistCalcArr, uint64_t * uniqueCellArr, uint64_t * cellNumPointsArr, unsigned int * nCells, unsigned int * nNonEmptyCells, int *incrementors, unsigned int * nAdjCells)
{
	unsigned int tid=threadIdx.x+ (blockIdx.x*BLOCKSIZE); 

	unsigned int numDistCalcs = 0;
	int tempIdx;

	// make sure thread is within unique cell count
	if (tid>=*nNonEmptyCells){
		return;
	}

	// get value of cell linearID for this thread
	uint64_t linearID = uniqueCellArr[tid];

	unsigned int pointIdx[NUMINDEXEDDIM];

	// get index of grid point
	getNDimIndexesFromLinearIdxGPU(pointIdx, nCells, NUMINDEXEDDIM, linearID);

	// check that linear ID does not already have adjacent cells caluclated for it
	unsigned int tempAdjCellIdx[NUMINDEXEDDIM];

	// loop through each incrementor vector
	for (int i=0; i<(*nAdjCells); i++)
	{
		// loop thhrough each entry
		for (int j=0; j<NUMINDEXEDDIM; j++)
		{
			// get adjacent cell index
			tempAdjCellIdx[j] = pointIdx[j] + incrementors[(i * NUMINDEXEDDIM) + j];
		}
		// convert index back to cell id
		uint64_t adjLinearID=getLinearID_nDimensionsGPU(tempAdjCellIdx, nCells, NUMINDEXEDDIM);

		// test if cell is nonempty
		tempIdx = binarySearch(uniqueCellArr, *nNonEmptyCells, adjLinearID);
		if(tempIdx != -1)
		{
			// add number of points in the cell to num distance calcs
			numDistCalcs += cellNumPointsArr[tempIdx];
		}
	}

	// set number of dist calcs
	celllDistCalcArr[tid] = numDistCalcs;
}


__global__ void kernelMapPointToNumDistCalcs(uint64_t * pointDistCalcArr, DTYPE* database, unsigned int *N, DTYPE* epsilon, DTYPE* minArr, unsigned int * nCells, uint64_t *cellDistCalcArr,  uint64_t * uniqueCellArr, unsigned int * nNonEmptyCells)
{
	unsigned int tid=threadIdx.x+ (blockIdx.x*BLOCKSIZE); 
	int tempIdx;

	if (tid>=*N){
		return;
	}

	unsigned int pointID=tid*(GPUNUMDIM);

	unsigned int tmpNDCellIdx[NUMINDEXEDDIM];
	for (int j=0; j<NUMINDEXEDDIM; j++){
		tmpNDCellIdx[j]=((database[pointID+j]-minArr[j])/(*epsilon));
	}
	uint64_t linearID=getLinearID_nDimensionsGPU(tmpNDCellIdx, nCells, NUMINDEXEDDIM);

	// get the index of linear ID in unique cell array
	tempIdx = binarySearch(uniqueCellArr, *nNonEmptyCells, linearID);
	if(tempIdx != -1)
	{
		// set value to number of distance calculations for that cell
		pointDistCalcArr[tid] = cellDistCalcArr[tempIdx];
	}
	else
	{
		// unexpected that point cell is not found
		printf("\nWARNING: The cell for point %d was not found\n", pointID);
	}
}

__global__ void kernelPairwiseDatabaseRotation( DTYPE * database, unsigned int *N, DTYPE * theta, unsigned int * dimPair ) {
	unsigned int tid=threadIdx.x+ (blockIdx.x*BLOCKSIZE);

	if (tid>=*N){
		return;
	}

	unsigned int pointID=tid*(GPUNUMDIM);

	DTYPE dim_0 = database[pointID + dimPair[0]];
	DTYPE dim_1 = database[pointID + dimPair[1]];

	DTYPE new_dim_0 = (dim_0 * cos(*theta)) - (dim_1 * sin(*theta));
	DTYPE new_dim_1 = (dim_0 * sin(*theta)) + (dim_1 * cos(*theta));

	database[pointID + dimPair[0]] = new_dim_0;
	database[pointID + dimPair[1]] = new_dim_1;
}